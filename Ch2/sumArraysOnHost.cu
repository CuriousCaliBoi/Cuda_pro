
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>

void sumArraysonHost (float *A, float *B, float *C,const int N){
    for (int idx = 0; idx < N; idx++){
        C[idx] = A[idx] + B[idx];
    }
}

// So you pass in I'm assuming float pointers probably to some array
// then we just add up all the respective values on A and B N times

// cool 😎

void initialData(float *ip, int size){
    // generate different seed for random number
    time_t t;
    srand((unsigned int) time(&t));
    // what the fuck is going on lol
    for (int i =0; i< size;i++){
        ip[i] = (float)(rand() & 0xFF )/10.0f;
    }
}

int main(int argc, char **argv) {
    int nElem = 1024;
    size_t nBytes = nElem * sizeof(float);

    float *h_A, *h_B,  *h_C;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    h_C = (float *)malloc(nBytes);

    initialData(h_A, nElem);
    initialData(h_B, nElem);

    sumArraysonHost(h_A, h_B, h_C, nElem);

    free(h_A);
    free(h_B);
    free(h_B);

    return(0);

}